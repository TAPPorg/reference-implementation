#include "../src/tapp/datatype.h"
#include "cutensor_bind.h"

hiptensorDataType_t translate_datatype(TAPP_datatype type)
{
    switch (type)
    {
    case TAPP_F32:
        return HIPTENSOR_R_32F;
        break;
    case TAPP_F64:
        return HIPTENSOR_R_64F;
        break;
    case TAPP_C32:
        return HIPTENSOR_C_32F;
        break;
    case TAPP_C64:
        return HIPTENSOR_C_64F;
        break;
    case TAPP_F16:
        return HIPTENSOR_R_16F;
        break;
    case TAPP_BF16:
        return HIPTENSOR_R_16BF;
        break;
    default: // TODO: Default should probably be an error
        return HIPTENSOR_R_32F;
        break;
    }
}

hiptensorComputeDescriptor_t translate_prectype(TAPP_prectype prec)
{
    switch (prec)
    {
        case TAPP_DEFAULT_PREC: // TODO: Make dependent on datatype
            return HIPTENSOR_COMPUTE_DESC_32F;
            break;
        case TAPP_F32F32_ACCUM_F32:
            return HIPTENSOR_COMPUTE_DESC_32F;
            break;
        case TAPP_F64F64_ACCUM_F64:
        return HIPTENSOR_COMPUTE_DESC_64F;
        case TAPP_F16F16_ACCUM_F16:
            return HIPTENSOR_COMPUTE_DESC_16F;
            break;
        default: // TODO: Default should probably be an error
            return HIPTENSOR_COMPUTE_DESC_32F;
            break;
    }
}

size_t sizeof_datatype(TAPP_datatype type)
{
    switch (type)
    {
    case TAPP_F32:
        return sizeof(float);
        break;
    case TAPP_F64:
        return sizeof(double);
        break;
    case TAPP_C32: 
        return sizeof(std::complex<float>);
        break;
    case TAPP_C64:
        return sizeof(std::complex<double>);
        break;
    /*case TAPP_F16: // Fix these datatypes
        //return _Float16;
        break;
    case TAPP_BF16:
        //return __bf16;
        break;*/
    default: // TODO: Default should probably be an error
        return sizeof(float);
        break;
    }
}