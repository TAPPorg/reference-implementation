#include "cutensor_bind.h"
#include "../src/tapp/handle.h"

TAPP_EXPORT TAPP_error create_handle(TAPP_handle* handle)//TAPP_error create_TAPP_handle(TAPP_handle* handle)
{
    hiptensorHandle_t* cuhandle = new hiptensorHandle_t;
    hiptensorCreate(cuhandle);
    *handle = (TAPP_handle) cuhandle;
    return 0; // TODO: implement cutensor error handling
}

TAPP_EXPORT TAPP_error TAPP_destroy_handle(TAPP_handle handle)
{
    hiptensorHandle_t* cuhandle = (hiptensorHandle_t*) handle;
    hiptensorDestroy(*cuhandle);
    delete cuhandle;
    return 0; // TODO: implement cutensor error handling
}