#include "../src/tapp/product.h"
#include "cutensor_bind.h"

hiptensorOperator_t translate_operator(TAPP_element_op op)
{
    switch (op)
    {
    case TAPP_IDENTITY:
        return HIPTENSOR_OP_IDENTITY;
        break;
    case TAPP_CONJUGATE:
        return HIPTENSOR_OP_CONJ;
        break;
    default: // TODO: Default should probably be an error
        return HIPTENSOR_OP_IDENTITY;
        break;
    }
}

TAPP_EXPORT TAPP_error TAPP_create_tensor_product(TAPP_tensor_product* plan,
                                                  TAPP_handle handle,
                                                  TAPP_element_op op_A,
                                                  TAPP_tensor_info A,
                                                  const int64_t* idx_A,
                                                  TAPP_element_op op_B,
                                                  TAPP_tensor_info B,
                                                  const int64_t* idx_B,
                                                  TAPP_element_op op_C,
                                                  TAPP_tensor_info C,
                                                  const int64_t* idx_C,
                                                  TAPP_element_op op_D,
                                                  TAPP_tensor_info D,
                                                  const int64_t* idx_D,
                                                  TAPP_prectype prec)
{
    cutensor_plan* cuplan = new cutensor_plan;
    hiptensorHandle_t cuhandle = *((hiptensorHandle_t*) handle);
    std::vector<int32_t> cuidx_A = std::vector<int32_t>(idx_A, idx_A + TAPP_get_nmodes(A));
    std::vector<int32_t> cuidx_B = std::vector<int32_t>(idx_B, idx_B + TAPP_get_nmodes(B));
    std::vector<int32_t> cuidx_C = std::vector<int32_t>(idx_C, idx_C + TAPP_get_nmodes(C));
    std::vector<int32_t> cuidx_D = std::vector<int32_t>(idx_D, idx_D + TAPP_get_nmodes(D));
    hiptensorOperationDescriptor_t desc;
    HANDLE_ERROR(hiptensorCreateContraction(cuhandle, 
                &desc,
                *((cutensor_info*)A)->desc, cuidx_A.data(), translate_operator(op_A),
                *((cutensor_info*)B)->desc, cuidx_B.data(), translate_operator(op_B),
                *((cutensor_info*)C)->desc, cuidx_C.data(), translate_operator(op_C),
                *((cutensor_info*)D)->desc, cuidx_D.data(),
                translate_prectype(prec)));

    hiptensorDataType_t scalarType;
    HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(cuhandle,
                desc,
                HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
                (void*)&scalarType,
                sizeof(scalarType)));

    assert(scalarType == HIPTENSOR_R_32F);

    const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    hiptensorPlanPreference_t planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(
                cuhandle,
                &planPref,
                algo,
                HIPTENSOR_JIT_MODE_NONE));

    uint64_t workspaceSizeEstimate = 0;
    const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
    hiptensorEstimateWorkspaceSize(cuhandle,
                desc,
                planPref,
                workspacePref,
                &workspaceSizeEstimate);

    cuplan->plan = new hiptensorPlan_t;
    HANDLE_ERROR(hiptensorCreatePlan(cuhandle,
                cuplan->plan,
                desc,
                planPref,
                workspaceSizeEstimate));
    cuplan->sizeA = ((cutensor_info*)A)->size;
    cuplan->sizeB = ((cutensor_info*)B)->size;
    cuplan->sizeC = ((cutensor_info*)C)->size;
    cuplan->sizeD = ((cutensor_info*)D)->size;
    *plan = (TAPP_tensor_product) cuplan;
    HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
    hiptensorDestroyPlanPreference(planPref);
    return 0; // TODO: implement cutensor error handling
}

TAPP_EXPORT TAPP_error TAPP_destroy_tensor_product(TAPP_tensor_product plan)
{
    cutensor_plan* cuplan = (cutensor_plan*) plan;
    HANDLE_ERROR(hiptensorDestroyPlan(*cuplan->plan));
    delete cuplan->plan;
    delete cuplan;
    return 0; // TODO: implement cutensor error handling
}
 
//TODO: in-place operation: set C = NULL or TAPP_IN_PLACE?
 
TAPP_EXPORT TAPP_error TAPP_execute_product(TAPP_tensor_product plan,
                                            TAPP_executor exec,
                                            TAPP_status* status,
                                            const void* alpha,
                                            const void* A,
                                            const void* B,
                                            const void* beta,
                                            const void* C,
                                                  void* D)
{    
    void *A_d, *B_d, *C_d, *D_d;
    hipMalloc((void**)&A_d, ((cutensor_plan*)plan)->sizeA);
    hipMalloc((void**)&B_d, ((cutensor_plan*)plan)->sizeB);
    hipMalloc((void**)&C_d, ((cutensor_plan*)plan)->sizeC);
    hipMalloc((void**)&D_d, ((cutensor_plan*)plan)->sizeD);
    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, ((cutensor_plan*)plan)->sizeA, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, ((cutensor_plan*)plan)->sizeB, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, ((cutensor_plan*)plan)->sizeC, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(D_d, D, ((cutensor_plan*)plan)->sizeD, hipMemcpyHostToDevice));
    assert(uintptr_t(A_d) % 128 == 0);
    assert(uintptr_t(B_d) % 128 == 0);
    assert(uintptr_t(C_d) % 128 == 0);
    assert(uintptr_t(D_d) % 128 == 0);
    hiptensorHandle_t handle;
    hiptensorCreate(&handle);
    hiptensorPlan_t* cuplan = ((cutensor_plan*) plan)->plan;
    uint64_t actualWorkspaceSize = 0;
    HANDLE_ERROR(hiptensorPlanGetAttribute(handle,
                *cuplan,
                HIPTENSOR_PLAN_REQUIRED_WORKSPACE,
                &actualWorkspaceSize,
                sizeof(actualWorkspaceSize)));

    void *work = nullptr;
    if (actualWorkspaceSize > 0)
    {
        HANDLE_CUDA_ERROR(hipMalloc(&work, actualWorkspaceSize));
        assert(uintptr_t(work) % 128 == 0);
    }
    hipStream_t stream;
    HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

    HANDLE_ERROR(hiptensorContract(handle,
                *cuplan,
                alpha, A_d, B_d,
                beta,  C_d, D_d, 
                work, actualWorkspaceSize, stream));

    HANDLE_CUDA_ERROR(hipStreamSynchronize(stream));
    HANDLE_CUDA_ERROR(hipMemcpy((void*) D, D_d, ((cutensor_plan*)plan)->sizeD, hipMemcpyDeviceToHost));

    hiptensorDestroy(handle);
    hipStreamDestroy(stream);

    if (A_d) hipFree(A_d);
    if (B_d) hipFree(B_d);
    if (C_d) hipFree(C_d);
    if (D_d) hipFree(D_d);
    if (work) hipFree(work);
    return 0; // TODO: implement cutensor error handling
}