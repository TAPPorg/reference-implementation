#include "../src/tapp/tensor.h"
#include "cutensor_bind.h"

TAPP_EXPORT TAPP_error TAPP_create_tensor_info(TAPP_tensor_info* info,
                                               TAPP_datatype type,
                                               int nmode,
                                               const int64_t* extents,
                                               const int64_t* strides)
{
    hiptensorHandle_t handle;
    hiptensorCreate(&handle);
    cutensor_info* tensor_info = new cutensor_info;
    tensor_info->desc = new hiptensorTensorDescriptor_t;
    const uint32_t kAlignment = 128;
    hiptensorCreateTensorDescriptor(handle,
                tensor_info->desc,
                nmode,
                extents,
                strides,
                translate_datatype(type), kAlignment);
    hiptensorDestroy(handle);
    size_t elements = 1;
    for (int i = 0; i < nmode; ++i)
        elements *= extents[i];
    tensor_info->copy_size = 1;
    tensor_info->data_offset = 0;
    for (int i = 0; i < nmode; i++)
    {
        tensor_info->copy_size += (extents[i] - 1)*strides[i];
        if (extents[i] < 0)
        {
            tensor_info->data_offset += extents[i] * strides[i];
        }
    }
    tensor_info->copy_size *= sizeof_datatype(type);
    tensor_info->data_offset *= sizeof_datatype(type);
    tensor_info->type = type;
    tensor_info->elements = elements;
    tensor_info->nmode = nmode;
    tensor_info->extents = new int64_t[nmode];
    tensor_info->strides = new int64_t[nmode];
    for (int i = 0; i < nmode; ++i)
    {
        tensor_info->extents[i] = extents[i];
        tensor_info->strides[i] = strides[i];
    }
    *info = (TAPP_tensor_info) tensor_info;
    return 0; // TODO: implement cutensor error handling
}

TAPP_EXPORT TAPP_error TAPP_destroy_tensor_info(TAPP_tensor_info info)
{
    cutensor_info* tensor_info = (cutensor_info*) info;
    hiptensorDestroyTensorDescriptor(*tensor_info->desc);
    delete tensor_info->desc;
    delete[] tensor_info->extents;
    delete[] tensor_info->strides;
    delete tensor_info;
    return 0; // TODO: implement cutensor error handling
}

TAPP_EXPORT int TAPP_get_nmodes(TAPP_tensor_info info)
{
    return ((cutensor_info*) info)->nmode;
}

TAPP_EXPORT TAPP_error TAPP_set_nmodes(TAPP_tensor_info info,
                                       int nmodes)
{
    return 0; // TODO: correctly implement, currently placeholder
}

TAPP_EXPORT void TAPP_get_extents(TAPP_tensor_info info,
                                  int64_t* extents)
{
    memcpy(extents, ((cutensor_info*) info)->extents, ((cutensor_info*) info)->nmode * sizeof(int64_t));
    return; // TODO: correctly implement, currently placeholder
}

TAPP_EXPORT TAPP_error TAPP_set_extents(TAPP_tensor_info info,
                                        const int64_t* extents)
{
    return 0; // TODO: correctly implement, currently placeholder
}

TAPP_EXPORT void TAPP_get_strides(TAPP_tensor_info info,
                                  int64_t* strides)
{
    memcpy(strides, ((cutensor_info*) info)->strides, ((cutensor_info*) info)->nmode * sizeof(int64_t));
    return; // TODO: correctly implement, currently placeholder
}

TAPP_EXPORT TAPP_error TAPP_set_strides(TAPP_tensor_info info,
                                        const int64_t* strides)
{
    return 0; // TODO: correctly implement, currently placeholder
}